#include "hip/hip_runtime.h"
#include "PCFG.h"
#include <cstring>
#include <vector>
#include <string>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>

// 优化后的kernel：用offsets和all_data定位每个字符串
__global__ void generate_guesses_kernel(const char *all_data, const int *offsets, int num, char *d_output, int max_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num) {
        int start = offsets[idx];
        int len = offsets[idx + 1] - start;
        const char *src = all_data + start;
        char *dst = d_output + idx * max_len;

        // 使用memcpy替代手动循环
        if (len > 0) {
            memcpy(dst, src, len);
        }
        dst[len] = '\0';
    }
}

// 可选：向量化拷贝（仅当max_len和src对齐时才有意义）
// __global__ void generate_guesses_kernel_vec(const char *all_data, const int *offsets, int num, char *d_output, int max_len) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < num) {
//         int start = offsets[idx];
//         int len = offsets[idx + 1] - start;
//         const char *src = all_data + start;
//         char *dst = d_output + idx * max_len;
//         int i = 0;
//         for (; i + 3 < len; i += 4) {
//             *(reinterpret_cast<int*>(dst + i)) = *(reinterpret_cast<const int*>(src + i));
//         }
//         for (; i < len; ++i) dst[i] = src[i];
//         dst[len] = '\0';
//     }
// }
#endif

void PriorityQueue::CalProb(PT &pt)
{
    pt.prob = pt.preterm_prob;
    int index = 0;

    for (int idx : pt.curr_indices)
    {
        if (pt.content[index].type == 1)
        {
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
        }
        index += 1;
    }
}

void PriorityQueue::init()
{
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        CalProb(pt);
        priority.emplace_back(pt); // 修正
    }
}

void PriorityQueue::PopNext()
{
    // 适配std::vector
    if (priority.empty()) return;
    Generate(priority.front());
    PT pt_top = priority.front(); // 先拷贝一份
    vector<PT> new_pts = pt_top.NewPTs();
    priority.erase(priority.begin());
    for (PT pt : new_pts)
    {
        CalProb(pt);
        priority.emplace_back(pt); // 修正
    }
}

void PriorityQueue::PopNextParallel() 
{
    int batch_size = 100;
    if (priority.empty()) return;
    int n = std::min(batch_size, (int)priority.size());
    std::vector<PT> batch_pts(priority.begin(), priority.begin() + n);

    int max_len = 0;
    std::vector<std::string> all_guesses;
    std::vector<int> offsets;
    offsets.push_back(0);

    for (PT& pt : batch_pts) {
        // 计算概率
        CalProb(pt);

        segment *a;
        std::string guess_prefix;
        int num;
        if (pt.content.size() == 1) {
            if (pt.content[0].type == 1)
                a = &m.letters[m.FindLetter(pt.content[0])];
            else if (pt.content[0].type == 2)
                a = &m.digits[m.FindDigit(pt.content[0])];
            else
                a = &m.symbols[m.FindSymbol(pt.content[0])];
            num = pt.max_indices[0];
            for (int i = 0; i < num; ++i) {
                const std::string& s = a->ordered_values[i];
                all_guesses.push_back(s);
                offsets.push_back(offsets.back() + s.size());
                if (s.size() > max_len) max_len = s.size();
            }
        } else {
            int seg_idx = 0;
            for (int idx : pt.curr_indices) {
                if (pt.content[seg_idx].type == 1)
                    guess_prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
                else if (pt.content[seg_idx].type == 2)
                    guess_prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
                else
                    guess_prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
                seg_idx += 1;
                if (seg_idx == pt.content.size() - 1)
                    break;
            }
            if (pt.content[pt.content.size() - 1].type == 1)
                a = &m.letters[m.FindLetter(pt.content[pt.content.size() - 1])];
            else if (pt.content[pt.content.size() - 1].type == 2)
                a = &m.digits[m.FindDigit(pt.content[pt.content.size() - 1])];
            else
                a = &m.symbols[m.FindSymbol(pt.content[pt.content.size() - 1])];
            num = pt.max_indices[pt.content.size() - 1];
            for (int i = 0; i < num; ++i) {
                std::string temp = guess_prefix + a->ordered_values[i];
                all_guesses.push_back(temp);
                offsets.push_back(offsets.back() + temp.size());
                if (temp.size() > max_len) max_len = temp.size();
            }
        }
    }
    int total_guesses = all_guesses.size();
    max_len += 1;

#ifdef __HIPCC__
    if (total_guesses > 10000) {
        // 拼接所有字符串到一块连续内存
        std::vector<char> all_data(offsets.back());
        int pos = 0;
        for (const auto& s : all_guesses) {
            memcpy(&all_data[pos], s.data(), s.size());
            pos += s.size();
        }

        static char *d_all_data = nullptr;
        static int *d_offsets = nullptr;
        static char *d_output = nullptr;
        static size_t all_data_capacity = 0, offsets_capacity = 0, output_capacity = 0;
        static hipStream_t stream = nullptr;
        if (!stream) hipStreamCreate(&stream);

        if (all_data.size() > all_data_capacity) {
            if (d_all_data) hipFree(d_all_data);
            hipMalloc(&d_all_data, all_data.size() * sizeof(char));
            all_data_capacity = all_data.size();
        }
        if (offsets.size() > offsets_capacity) {
            if (d_offsets) hipFree(d_offsets);
            hipMalloc(&d_offsets, offsets.size() * sizeof(int));
            offsets_capacity = offsets.size();
        }
        if ((total_guesses * max_len) > output_capacity) {
            if (d_output) hipFree(d_output);
            hipMalloc(&d_output, total_guesses * max_len * sizeof(char));
            output_capacity = total_guesses * max_len;
        }

        hipMemcpyAsync(d_all_data, all_data.data(), all_data.size() * sizeof(char), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice, stream);

        int block = 256;
        int grid = (total_guesses + block - 1) / block;
        generate_guesses_kernel<<<grid, block, 0, stream>>>(d_all_data, d_offsets, total_guesses, d_output, max_len);

        static std::vector<char> h_output;
        h_output.resize(total_guesses * max_len);
        hipMemcpyAsync(h_output.data(), d_output, total_guesses * max_len * sizeof(char), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        guesses.reserve(guesses.size() + total_guesses);
        for (int i = 0; i < total_guesses; ++i) {
            guesses.emplace_back(&h_output[i * max_len]);
            this->total_guesses += 1;
        }
    } else
#endif
    {
        // CPU串行处理
        for (const auto& s : all_guesses) {
            guesses.emplace_back(s);
            this->total_guesses += 1;
        }
    }

    // 生成新PT并加入队列
    for (PT &pt : batch_pts) {
        vector<PT> new_pts = pt.NewPTs();
        for (PT &new_pt : new_pts) {
            CalProb(new_pt);
            priority.emplace_back(new_pt);
        }
    }
    priority.erase(priority.begin(), priority.begin() + n);
}

vector<PT> PT::NewPTs()
{
    vector<PT> res;
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        int init_pivot = pivot;
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            curr_indices[i] += 1;
            if (curr_indices[i] < max_indices[i])
            {
                pivot = i;
                res.emplace_back(*this);
            }
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }
    return res;
}


void PriorityQueue::Generate(PT pt)
{
    CalProb(pt);

    if (pt.content.size() == 1)
    {
        segment *a;
        if (pt.content[0].type == 1)
            a = &m.letters[m.FindLetter(pt.content[0])];
        if (pt.content[0].type == 2)
            a = &m.digits[m.FindDigit(pt.content[0])];
        if (pt.content[0].type == 3)
            a = &m.symbols[m.FindSymbol(pt.content[0])];

        int num = pt.max_indices[0];
        int max_len = 0;
        for (int i = 0; i < num; ++i)
            if (a->ordered_values[i].size() > max_len)
                max_len = a->ordered_values[i].size();
        max_len += 1;
        //std::cout<<num<<endl;
        // 判断任务量，选择CPU或GPU
        if (num <= 1000000) {
            // CPU串行实现（参考guessing.cpp）
            for (int i = 0; i < num; i += 1)
            {
                string guess = a->ordered_values[i];
                guesses.emplace_back(guess);
                total_guesses += 1;
            }
        }
#ifdef __HIPCC__
        else
        {
            std::cout<<"GPU"<<endl;
            // 静态内存复用（仅示例，线程安全需注意）
            static char *d_all_data = nullptr;
            static int *d_offsets = nullptr;
            static char *d_output = nullptr;
            static size_t all_data_capacity = 0, offsets_capacity = 0, output_capacity = 0;
            static hipStream_t stream = nullptr;
            if (!stream) hipStreamCreate(&stream);

            // 拼接所有字符串到一块连续内存，并记录offsets
            std::vector<char> all_data;
            std::vector<int> offsets(num + 1);
            int pos = 0;
            for (int i = 0; i < num; ++i) {
                offsets[i] = pos;
                const std::string &s = a->ordered_values[i];
                all_data.insert(all_data.end(), s.begin(), s.end());
                pos += s.size();
            }
            offsets[num] = pos;

            // 分配或复用 device 内存
            if (all_data.size() > all_data_capacity) {
                if (d_all_data) hipFree(d_all_data);
                hipMalloc(&d_all_data, all_data.size() * sizeof(char));
                all_data_capacity = all_data.size();
            }
            if ((num + 1) > offsets_capacity) {
                if (d_offsets) hipFree(d_offsets);
                hipMalloc(&d_offsets, (num + 1) * sizeof(int));
                offsets_capacity = num + 1;
            }
            if ((num * max_len) > output_capacity) {
                if (d_output) hipFree(d_output);
                hipMalloc(&d_output, num * max_len * sizeof(char));
                output_capacity = num * max_len;
            }

            // 异步拷贝
            hipMemcpyAsync(d_all_data, all_data.data(), all_data.size() * sizeof(char), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_offsets, offsets.data(), (num + 1) * sizeof(int), hipMemcpyHostToDevice, stream);

            int block = 256;
            int grid = (num + block - 1) / block;
            generate_guesses_kernel<<<grid, block, 0, stream>>>(d_all_data, d_offsets, num, d_output, max_len);

            // 异步拷贝回主机
            static std::vector<char> h_output;
            h_output.resize(num * max_len);
            hipMemcpyAsync(h_output.data(), d_output, num * max_len * sizeof(char), hipMemcpyDeviceToHost, stream);

            // 只在需要时同步
            hipStreamSynchronize(stream);

            guesses.reserve(guesses.size() + num);
            for (int i = 0; i < num; ++i) {
                guesses.emplace_back(&h_output[i * max_len]);
                total_guesses += 1;
            }
        }
#endif
    }
    else
    {
        string guess_prefix;
        int seg_idx = 0;
        for (int idx : pt.curr_indices)
        {
            if (pt.content[seg_idx].type == 1)
                guess_prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            if (pt.content[seg_idx].type == 2)
                guess_prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            if (pt.content[seg_idx].type == 3)
                guess_prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            seg_idx += 1;
            if (seg_idx == pt.content.size() - 1)
                break;
        }

        segment *a;
        if (pt.content[pt.content.size() - 1].type == 1)
            a = &m.letters[m.FindLetter(pt.content[pt.content.size() - 1])];
        if (pt.content[pt.content.size() - 1].type == 2)
            a = &m.digits[m.FindDigit(pt.content[pt.content.size() - 1])];
        if (pt.content[pt.content.size() - 1].type == 3)
            a = &m.symbols[m.FindSymbol(pt.content[pt.content.size() - 1])];

        int num = pt.max_indices[pt.content.size() - 1];
        int max_len = guess_prefix.size();
        for (int i = 0; i < num; ++i)
            if (a->ordered_values[i].size() + guess_prefix.size() > max_len)
                max_len = a->ordered_values[i].size() + guess_prefix.size();
        max_len += 1;
        //std::cout<<num<<endl;
        // 判断任务量，选择CPU或GPU
        if (num <= 1000000) {
            // CPU串行实现（参考guessing.cpp）
            for (int i = 0; i < num; i += 1)
            {
                string temp = guess_prefix + a->ordered_values[i];
                guesses.emplace_back(temp);
                total_guesses += 1;
            }
        }
#ifdef __HIPCC__
        else
        {
            std::cout<<"GPU"<<endl;
            // 拼接所有字符串到一块连续内存，并记录offsets
            std::vector<char> all_data;
            std::vector<int> offsets(num + 1);
            int pos = 0;
            for (int i = 0; i < num; ++i) {
                std::string temp = guess_prefix + a->ordered_values[i];
                offsets[i] = pos;
                all_data.insert(all_data.end(), temp.begin(), temp.end());
                pos += temp.size();
            }
            offsets[num] = pos;

            // 静态内存复用（仅示例，线程安全需注意）
            static char *d_all_data = nullptr;
            static int *d_offsets = nullptr;
            static char *d_output = nullptr;
            static size_t all_data_capacity = 0, offsets_capacity = 0, output_capacity = 0;
            static hipStream_t stream = nullptr;
            if (!stream) hipStreamCreate(&stream);

            // 分配或复用 device 内存
            if (all_data.size() > all_data_capacity) {
                if (d_all_data) hipFree(d_all_data);
                hipMalloc(&d_all_data, all_data.size() * sizeof(char));
                all_data_capacity = all_data.size();
            }
            if ((num + 1) > offsets_capacity) {
                if (d_offsets) hipFree(d_offsets);
                hipMalloc(&d_offsets, (num + 1) * sizeof(int));
                offsets_capacity = num + 1;
            }
            if ((num * max_len) > output_capacity) {
                if (d_output) hipFree(d_output);
                hipMalloc(&d_output, num * max_len * sizeof(char));
                output_capacity = num * max_len;
            }

            // 异步拷贝
            hipMemcpyAsync(d_all_data, all_data.data(), all_data.size() * sizeof(char), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_offsets, offsets.data(), (num + 1) * sizeof(int), hipMemcpyHostToDevice, stream);

            int block = 256;
            int grid = (num + block - 1) / block;
            generate_guesses_kernel<<<grid, block, 0, stream>>>(d_all_data, d_offsets, num, d_output, max_len);

            // 异步拷贝回主机
            static std::vector<char> h_output;
            h_output.resize(num * max_len);
            hipMemcpyAsync(h_output.data(), d_output, num * max_len * sizeof(char), hipMemcpyDeviceToHost, stream);

            // 只在需要时同步
            hipStreamSynchronize(stream);

            guesses.reserve(guesses.size() + num);
            for (int i = 0; i < num; ++i) {
                guesses.emplace_back(&h_output[i * max_len]);
                total_guesses += 1;
            }
        }
#endif
    }
}