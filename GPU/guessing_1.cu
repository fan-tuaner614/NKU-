#include "hip/hip_runtime.h"
#include "PCFG.h"
#include <cstring>
#include <vector>
#include <string>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>

// 优化后的kernel：用offsets和all_data定位每个字符串
__global__ void generate_guesses_kernel(const char *all_data, const int *offsets, int num, char *d_output, int max_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num) {
        int start = offsets[idx];
        int len = offsets[idx + 1] - start;
        const char *src = all_data + start;
        char *dst = d_output + idx * max_len;

        // 使用memcpy替代手动循环
        if (len > 0) {
            memcpy(dst, src, len);
        }
        dst[len] = '\0';
    }
}

// 可选：向量化拷贝（仅当max_len和src对齐时才有意义）
// __global__ void generate_guesses_kernel_vec(const char *all_data, const int *offsets, int num, char *d_output, int max_len) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < num) {
//         int start = offsets[idx];
//         int len = offsets[idx + 1] - start;
//         const char *src = all_data + start;
//         char *dst = d_output + idx * max_len;
//         int i = 0;
//         for (; i + 3 < len; i += 4) {
//             *(reinterpret_cast<int*>(dst + i)) = *(reinterpret_cast<const int*>(src + i));
//         }
//         for (; i < len; ++i) dst[i] = src[i];
//         dst[len] = '\0';
//     }
// }
#endif

void PriorityQueue::CalProb(PT &pt)
{
    pt.prob = pt.preterm_prob;
    int index = 0;

    for (int idx : pt.curr_indices)
    {
        if (pt.content[index].type == 1)
        {
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
        }
        index += 1;
    }
}

void PriorityQueue::init()
{
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        CalProb(pt);
        priority.emplace_back(pt); // 修正
    }
}

void PriorityQueue::PopNext()
{
    // 适配std::vector
    if (priority.empty()) return;
    Generate(priority.front());
    PT pt_top = priority.front(); // 先拷贝一份
    vector<PT> new_pts = pt_top.NewPTs();
    priority.erase(priority.begin());
    for (PT pt : new_pts)
    {
        CalProb(pt);
        priority.emplace_back(pt); // 修正
    }
}

vector<PT> PT::NewPTs()
{
    vector<PT> res;
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        int init_pivot = pivot;
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            curr_indices[i] += 1;
            if (curr_indices[i] < max_indices[i])
            {
                pivot = i;
                res.emplace_back(*this);
            }
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }
    return res;
}


void PriorityQueue::Generate(PT pt)
{
    CalProb(pt);

    if (pt.content.size() == 1)
    {
        segment *a;
        if (pt.content[0].type == 1)
            a = &m.letters[m.FindLetter(pt.content[0])];
        if (pt.content[0].type == 2)
            a = &m.digits[m.FindDigit(pt.content[0])];
        if (pt.content[0].type == 3)
            a = &m.symbols[m.FindSymbol(pt.content[0])];

        int num = pt.max_indices[0];
        int max_len = 0;
        for (int i = 0; i < num; ++i)
            if (a->ordered_values[i].size() > max_len)
                max_len = a->ordered_values[i].size();
        max_len += 1;

#ifdef __HIPCC__
        // 静态内存复用（仅示例，线程安全需注意）
        static char *d_all_data = nullptr;
        static int *d_offsets = nullptr;
        static char *d_output = nullptr;
        static size_t all_data_capacity = 0, offsets_capacity = 0, output_capacity = 0;
        static hipStream_t stream = nullptr;
        if (!stream) hipStreamCreate(&stream);

        // 拼接所有字符串到一块连续内存，并记录offsets
        std::vector<char> all_data;
        std::vector<int> offsets(num + 1);
        int pos = 0;
        for (int i = 0; i < num; ++i) {
            offsets[i] = pos;
            const std::string &s = a->ordered_values[i];
            all_data.insert(all_data.end(), s.begin(), s.end());
            pos += s.size();
        }
        offsets[num] = pos;

        // 分配或复用 device 内存
        if (all_data.size() > all_data_capacity) {
            if (d_all_data) hipFree(d_all_data);
            hipMalloc(&d_all_data, all_data.size() * sizeof(char));
            all_data_capacity = all_data.size();
        }
        if ((num + 1) > offsets_capacity) {
            if (d_offsets) hipFree(d_offsets);
            hipMalloc(&d_offsets, (num + 1) * sizeof(int));
            offsets_capacity = num + 1;
        }
        if ((num * max_len) > output_capacity) {
            if (d_output) hipFree(d_output);
            hipMalloc(&d_output, num * max_len * sizeof(char));
            output_capacity = num * max_len;
        }

        // 异步拷贝
        hipMemcpyAsync(d_all_data, all_data.data(), all_data.size() * sizeof(char), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_offsets, offsets.data(), (num + 1) * sizeof(int), hipMemcpyHostToDevice, stream);

        int block = 256;
        int grid = (num + block - 1) / block;
        generate_guesses_kernel<<<grid, block, 0, stream>>>(d_all_data, d_offsets, num, d_output, max_len);

        // 异步拷贝回主机
        static std::vector<char> h_output;
        h_output.resize(num * max_len);
        hipMemcpyAsync(h_output.data(), d_output, num * max_len * sizeof(char), hipMemcpyDeviceToHost, stream);

        // 只在需要时同步
        hipStreamSynchronize(stream);

        guesses.reserve(guesses.size() + num);
        for (int i = 0; i < num; ++i) {
            guesses.emplace_back(&h_output[i * max_len]);
            total_guesses += 1;
        }
#else
        // 串行回退
        for (int i = 0; i < num; i += 1)
        {
            string guess = a->ordered_values[i];
            guesses.emplace_back(guess);
            total_guesses += 1;
        }
#endif
    }
    else
    {
        string guess_prefix;
        int seg_idx = 0;
        for (int idx : pt.curr_indices)
        {
            if (pt.content[seg_idx].type == 1)
                guess_prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            if (pt.content[seg_idx].type == 2)
                guess_prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            if (pt.content[seg_idx].type == 3)
                guess_prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            seg_idx += 1;
            if (seg_idx == pt.content.size() - 1)
                break;
        }

        segment *a;
        if (pt.content[pt.content.size() - 1].type == 1)
            a = &m.letters[m.FindLetter(pt.content[pt.content.size() - 1])];
        if (pt.content[pt.content.size() - 1].type == 2)
            a = &m.digits[m.FindDigit(pt.content[pt.content.size() - 1])];
        if (pt.content[pt.content.size() - 1].type == 3)
            a = &m.symbols[m.FindSymbol(pt.content[pt.content.size() - 1])];

        int num = pt.max_indices[pt.content.size() - 1];
        int max_len = guess_prefix.size();
        for (int i = 0; i < num; ++i)
            if (a->ordered_values[i].size() + guess_prefix.size() > max_len)
                max_len = a->ordered_values[i].size() + guess_prefix.size();
        max_len += 1;

#ifdef __HIPCC__
        // 拼接所有字符串到一块连续内存，并记录offsets
        std::vector<char> all_data;
        std::vector<int> offsets(num + 1);
        int pos = 0;
        for (int i = 0; i < num; ++i) {
            std::string temp = guess_prefix + a->ordered_values[i];
            offsets[i] = pos;
            all_data.insert(all_data.end(), temp.begin(), temp.end());
            pos += temp.size();
        }
        offsets[num] = pos;

        // 静态内存复用（仅示例，线程安全需注意）
        static char *d_all_data = nullptr;
        static int *d_offsets = nullptr;
        static char *d_output = nullptr;
        static size_t all_data_capacity = 0, offsets_capacity = 0, output_capacity = 0;
        static hipStream_t stream = nullptr;
        if (!stream) hipStreamCreate(&stream);

        // 分配或复用 device 内存
        if (all_data.size() > all_data_capacity) {
            if (d_all_data) hipFree(d_all_data);
            hipMalloc(&d_all_data, all_data.size() * sizeof(char));
            all_data_capacity = all_data.size();
        }
        if ((num + 1) > offsets_capacity) {
            if (d_offsets) hipFree(d_offsets);
            hipMalloc(&d_offsets, (num + 1) * sizeof(int));
            offsets_capacity = num + 1;
        }
        if ((num * max_len) > output_capacity) {
            if (d_output) hipFree(d_output);
            hipMalloc(&d_output, num * max_len * sizeof(char));
            output_capacity = num * max_len;
        }

        // 异步拷贝
        hipMemcpyAsync(d_all_data, all_data.data(), all_data.size() * sizeof(char), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_offsets, offsets.data(), (num + 1) * sizeof(int), hipMemcpyHostToDevice, stream);

        int block = 256;
        int grid = (num + block - 1) / block;
        generate_guesses_kernel<<<grid, block, 0, stream>>>(d_all_data, d_offsets, num, d_output, max_len);

        // 异步拷贝回主机
        static std::vector<char> h_output;
        h_output.resize(num * max_len);
        hipMemcpyAsync(h_output.data(), d_output, num * max_len * sizeof(char), hipMemcpyDeviceToHost, stream);

        // 只在需要时同步
        hipStreamSynchronize(stream);

        guesses.reserve(guesses.size() + num);
        for (int i = 0; i < num; ++i) {
            guesses.emplace_back(&h_output[i * max_len]);
            total_guesses += 1;
        }
#else
        // 串行回退
        for (int i = 0; i < num; i += 1)
        {
            string temp = guess_prefix + a->ordered_values[i];
            guesses.emplace_back(temp);
            total_guesses += 1;
        }
#endif
    }
}